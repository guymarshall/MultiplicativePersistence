#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include <vector>

#include "maths.cuh"

#define LOG(input) std::cout << input << std::endl;

// __global__ void add(uint64_t *a, uint64_t *b, uint64_t *c, size_t size) {
//     uint64_t i = threadIdx.x + blockDim.x * blockIdx.x;
//     if (i < size) {
//         c[i] = a[i] + b[i];
//     }
// }

int main(void) {
    uint8_t highestStepsCount = 0;
    uint64_t highestStepsNumber = 0;
    uint64_t finish = 277777788888899;

    for (uint64_t i = 0; i <= finish; i++) {
        uint16_t result = multiplicativePersistence(i);

        if (result > highestStepsCount) {
            highestStepsCount = result;
            highestStepsNumber = i;
            std::cout << "Up to " << i << " so far with " << result << " steps" << std::endl;
        }
    }

    std::cout << "Highest step count: " << highestStepsNumber << " at " << highestStepsCount << std::endl;




    // const uint64_t LIMIT = 1024;
    // const size_t size = LIMIT * sizeof(uint64_t);

    // std::vector<uint64_t> vectorA(LIMIT);
    // std::vector<uint64_t> vectorB(LIMIT);
    // std::vector<uint64_t> vectorC(LIMIT);

    // for (uint64_t i = 0; i < LIMIT; i++) {
    //     vectorA[i] = i;
    //     vectorB[i] = LIMIT - i;
    // }

    // uint64_t *deviceA, *deviceB, *deviceC;

    // hipMalloc(&deviceA, size);
    // hipMalloc(&deviceB, size);
    // hipMalloc(&deviceC, size);

    // hipMemcpy(deviceA, vectorA.data(), size, hipMemcpyHostToDevice);
    // hipMemcpy(deviceB, vectorB.data(), size, hipMemcpyHostToDevice);

    // add<<<LIMIT / 256, 256>>>(deviceA, deviceB, deviceC, LIMIT);

    // hipDeviceSynchronize();

    // hipMemcpy(vectorC.data(), deviceC, size, hipMemcpyDeviceToHost);

    // hipFree(deviceA);
    // hipFree(deviceB);
    // hipFree(deviceC);

    // uint64_t resultSum = 0;
    // for (uint64_t i = 0; i < LIMIT; i++) {
    //     resultSum += vectorC[i];
    // }

    // std::cout << "Result: sum = " << resultSum << std::endl;

    return 0;
}