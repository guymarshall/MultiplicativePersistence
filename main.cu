#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include "maths.cuh"

const uint16_t NUMBERS_PER_CHUNK = 1024;
const uint16_t CUDA_CORES = 896;
const uint64_t FINISH = 277777788888899;

__global__ void calculatePersistence(uint64_t chunkStart, uint8_t* stepCounts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < NUMBERS_PER_CHUNK) {
        uint64_t currentNumber = chunkStart + tid;
        stepCounts[tid] = multiplicativePersistence(currentNumber);
    }
}

int main() {
    uint8_t highestStepsCount = 0;
    uint64_t highestStepsNumber = 0;

    const size_t size = NUMBERS_PER_CHUNK * sizeof(uint8_t);
    uint8_t* hostStepCounts = (uint8_t*) malloc(size);

    uint8_t* deviceStepCounts;
    hipMalloc((void**) &deviceStepCounts, size);

    for (uint64_t chunkStart = 0; chunkStart <= FINISH - NUMBERS_PER_CHUNK; chunkStart += NUMBERS_PER_CHUNK) {
        calculatePersistence<<<1, CUDA_CORES>>>(chunkStart, deviceStepCounts);

        hipMemcpy(hostStepCounts, deviceStepCounts, size, hipMemcpyDeviceToHost);

        for (int i = 0; i < NUMBERS_PER_CHUNK; ++i) {
            uint64_t currentNumber = chunkStart + i;
            uint8_t result = hostStepCounts[i];

            if (result > highestStepsCount) {
                highestStepsCount = result;
                highestStepsNumber = currentNumber;
                printf("Up to %ld so far with %d steps\n", currentNumber, result);
            }
        }
    }

    printf("Highest step count: %ld at %d\n", highestStepsNumber, highestStepsCount);

    hipFree(deviceStepCounts);
    free(hostStepCounts);

    return 0;
}